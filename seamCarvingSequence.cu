#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <time.h>

struct CpuTimer
{
    clock_t start;
    clock_t end;

    CpuTimer()
    {
        // Constructor
    }

    ~CpuTimer()
    {
        // De-constructor
    }

    void Start()
    {
        start = clock();
    }

    void Stop()
    {
        end = clock();
    }

    double Elapsed()
    {
        return ((double) (end - start)) / CLOCKS_PER_SEC;
    }
};

struct EngeryPoint
{
    int val;        // Energy of current point.
    int prePos;     // Postion of pre enegry.
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
    FILE * f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);

    if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);

    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) // In this exercise, we assume 1 byte per value
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    for (int i = 0; i < width * height; i++)
        fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

    fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

char * concatStr(const char * s1, const char * s2)
{
	char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

void setValAndPostionEnergy(uint8_t * inPixels, EngeryPoint ** energyTable, int rowImg, int colImg, int width, int height)
{
    int energy_tmp = energyTable[rowImg + 1][colImg].val;
    int position_tmp = (rowImg + 1) * width + colImg;
    if (colImg - 1 >= 0)
    {
        if (energy_tmp > energyTable[rowImg + 1][colImg - 1].val)
        {
            energy_tmp = energyTable[rowImg + 1][colImg - 1].val;
            position_tmp = (rowImg + 1) * width + colImg - 1;
        }
    }
    if (colImg + 1 < height)
    {
        if (energy_tmp > energyTable[rowImg + 1][colImg + 1].val)
        {
            energy_tmp = energyTable[rowImg + 1][colImg + 1].val;
            position_tmp = (rowImg + 1) * width + colImg + 1;
        }
    }
    energyTable[rowImg][colImg].val = energy_tmp + inPixels[rowImg * width + colImg];
    energyTable[rowImg][colImg].prePos = position_tmp;
}

void convertRgb2Gray(uchar3 * inPixels, int width, int height, uint8_t * &outPixels)
{
    outPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue
    for (int rowImg = 0; rowImg < height; rowImg++)
    {
        for (int colImg = 0; colImg < width; colImg++)
        {
            int idx = rowImg * width + colImg;
            uint8_t red = inPixels[idx].x;
            uint8_t green = inPixels[idx].y;
            uint8_t blue = inPixels[idx].z;
            outPixels[idx] = 0.299f*red + 0.587f*green + 0.114f*blue;
        }
    }
}

void detectEdgeImg(uint8_t * inPixels, int width, int height, uint8_t * &outPixels)
{
    int filterWidth = 3;
    float xFilter[filterWidth * filterWidth] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
    float yFilter[filterWidth * filterWidth] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
    outPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));

    for (int rowImg = 0; rowImg < height; rowImg++)
    {
        for (int colImg = 0; colImg < width; colImg++)
        {
            float xSobel = 0;
            float ySobel = 0;
            for (int colFilter = 0; colFilter < filterWidth; colFilter++)
            {
                for (int rowFilter = 0; rowFilter < filterWidth; rowFilter++)
                {
                    float xFilterVal = xFilter[rowFilter * filterWidth + colFilter] / 4;
                    float yFilterVal = yFilter[rowFilter * filterWidth + colFilter] / 4;
                    int inRowImg = rowImg + rowFilter - filterWidth / 2;
                    int inColImg = colImg + colFilter - filterWidth / 2;
                    inRowImg = min(max(inRowImg, 0), height - 1);
                    inColImg = min(max(inColImg, 0), width - 1);
                    xSobel += xFilterVal * inPixels[inRowImg * width + inColImg];
                    ySobel += yFilterVal * inPixels[inRowImg * width + inColImg];
                }
            }

            outPixels[rowImg * width + colImg] = sqrt(xSobel * xSobel + ySobel * ySobel);
        }
    }
}

void findSeamCarving(uint8_t * inPixels, int width, int height, int * traces)
{
    EngeryPoint ** energyTable = (EngeryPoint **)malloc(height * sizeof(EngeryPoint *));
    for (int row = 0; row < height; row++)
    {
        energyTable[row] = (EngeryPoint *)malloc(width * sizeof(EngeryPoint));
    }
    for (int colImg = 0; colImg < width; colImg++)
    {
        energyTable[height - 1][colImg].val = inPixels[(height - 1) * width + colImg];
        energyTable[height - 1][colImg].prePos = -1;
    }
    for (int rowImg = height - 2; rowImg >= 0; rowImg--)
    {
        for (int colImg = 0; colImg < width; colImg++)
        {
            setValAndPostionEnergy(inPixels, energyTable, rowImg, colImg, width, height);
        }
    }

    int minEnergy = energyTable[0][0].val;
    int minPostion = 0;
    int index = 0;
    for (int col = 0; col < width; col++)
    {
        if (minEnergy > energyTable[0][col].val)
        {
            minEnergy = energyTable[0][col].val;
            minPostion = col;
        }
    }

    while (index < height)
    {
        traces[index] = minPostion;
        minPostion = energyTable[minPostion / width][minPostion % width].prePos;
        index++;
    }

    // free energy tables
    for (int row = 0; row < height; row++)
    {
        free(energyTable[row]);
    }
    free(energyTable);
}

void cutSeamCarvingImg(uint8_t * inPixels, int width, int height, int * traces)
{
    for (int row = height - 1; row >= 0; row--)
    {
        for (int idx = traces[row]; idx < width * height + height - 2 - row; idx++)
        {
            inPixels[idx] = inPixels[idx + 1];
        }
    }
}

void cutSeamCarvingRGBImg(uchar3 * inPixels, int width, int height, int * traces)
{
    for (int row = height - 1; row >= 0; row--)
    {
        for (int idx = traces[row]; idx < width * height - 1; idx++)
        {
            inPixels[idx].x = inPixels[idx + 1].x;
            inPixels[idx].y = inPixels[idx + 1].y;
            inPixels[idx].z = inPixels[idx + 1].z;
        }
    }
}

void seamCarvingImg(uchar3 * inPixels, int width, int height, uchar3 * &outPixels, int size)
{
    CpuTimer timer;
    timer.Start();
    int maxCol = width;
    int maxRow = height;
    uint8_t * grayOutPixels, * edgeOutPixels;
    int * traces = (int *)malloc(height * sizeof(int));
    convertRgb2Gray(inPixels, width, height, grayOutPixels);
    detectEdgeImg(grayOutPixels, width, height, edgeOutPixels);
    free(grayOutPixels);
    for (int loop = 0; loop < size; loop++)
    {
        findSeamCarving(edgeOutPixels, maxCol, maxRow, traces);
        cutSeamCarvingImg(edgeOutPixels, maxCol, maxRow, traces);
        cutSeamCarvingRGBImg(inPixels, maxCol, maxRow, traces);
        maxCol--;
    }

    outPixels = (uchar3 *)malloc((width - size) * height * sizeof(uchar3));
    for (int idx = 0; idx < maxCol * height; idx++)
    {
        outPixels[idx].x = inPixels[idx].x;
        outPixels[idx].y = inPixels[idx].y;
        outPixels[idx].z = inPixels[idx].z;
    }

    // Free memories
    free(traces);
    free(edgeOutPixels);
    timer.Stop();
    double time = timer.Elapsed();
    printf("Processing time (use host): %f s\n\n", time);
}

int main(int argc, char ** argv)
{
    if (argc != 3 && argc != 4)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
    }

    int size = 1;
    if (argc == 4)
    {
        size = atoi(argv[3]);
    }
    
    // Read input image file
	int width, height;
    uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
    printf("Image size (width x height): %i x %i\n\n", width, height);
    if (width < size)
    {
        printf("The width of image less than size!\n");
        return EXIT_FAILURE;
    }
    
    // Seam Carving input image using host
    uchar3 * seamCarvingOutPixels;
    seamCarvingImg(inPixels, width, height, seamCarvingOutPixels, size);

    // Write results to files
    char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
    writePnm(seamCarvingOutPixels, width - size, height, concatStr(outFileNameBase, "_host.pnm"));

    // Free memories
    free(seamCarvingOutPixels);
    free(inPixels);
}