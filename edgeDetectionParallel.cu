
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName,
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);

	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);

	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height,
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "P3\n%i\n%i\n255\n", width, height);

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);

	fclose(f);
}

__global__ void blurImgKernel(uchar3* inPixels, int width, int height,
		float * filter, int filterWidth,
		uchar3* outPixels)
{
	// TODO
  int r = threadIdx.x + blockIdx.x * blockDim.x;
	int c = threadIdx.y + blockIdx.y * blockDim.y;// chỉ số phần tử pixle
  if (r < width && c < height) //check có cần làm hay không, thread ngoài biên thì không làm
  {
    int i = r * width + c;
    float outx = 0.0f;
    float outy = 0.0f;
    float outz = 0.0f;
    int iFilter = 0; //chỉ số phần tử ở filter
    for (int x = -filterWidth/2; x <= filterWidth/2; x++)
      for (int y = -filterWidth/2; y <= filterWidth/2; y++) //duyệt filter
      {
        int rx = r + x;
        int cy = c + y;
        if (rx < 0) rx = 0;
        if (rx > width - 1) rx = width - 1;
        if (cy < 0) cy = 0;
        if (cy > height - 1) cy = height - 1; // ngoài biên thì lấy phần tử gần nhất
        int k = rx * width + cy;
        outx += inPixels[k].x * filter[iFilter];
        outy += inPixels[k].y * filter[iFilter];
        outz += inPixels[k].z * filter[iFilter];// nhân với filter rồi cộng lại
        iFilter++;
      }
    outPixels[i].x = (unsigned char)outx;
    outPixels[i].y = (unsigned char)outy;
    outPixels[i].z = (unsigned char)outz;
  }
}

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth,
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO
    for (int r = 0; r < width; r++)
      for (int c = 0; c < height; c++) //duyệt pixels
        {
          int i = r * width + c;
          float outx = 0.0f;
          float outy = 0.0f;
          float outz = 0.0f;
          int iFilter = 0; //chỉ số phần tử ở filter
          for (int x = -filterWidth/2; x <= filterWidth/2; x++)
            for (int y = -filterWidth/2; y <= filterWidth/2; y++) //duyệt filter
            {
              int rx = r + x;
              int cy = c + y;
              if (rx < 0) rx = 0;
              if (rx > width - 1) rx = width - 1;
              if (cy < 0) cy = 0;
              if (cy > height - 1) cy = height - 1; // ngoài biên thì lấy phần tử gần nhất
              int k = rx * width + cy;
              outx += inPixels[k].x * filter[iFilter];
              outy += inPixels[k].y * filter[iFilter];
              outz += inPixels[k].z * filter[iFilter];// nhân với filter rồi cộng lại
              iFilter++;
            }
          outPixels[i].x = (unsigned char)outx;
          outPixels[i].y = (unsigned char)outy;
          outPixels[i].z = (unsigned char)outz;
        }
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO
    //Host allocates memories on device
    uchar3 *d_inPixels, *d_outPixels;
    float *d_filter;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    CHECK(hipMalloc(&d_outPixels, width * height * sizeof(uchar3)));
    CHECK(hipMalloc(&d_filter, filterWidth * filterWidth * sizeof(float)));
    //Host copies data to divece memories
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_filter, filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice));
    //Host invokes kernel function
    dim3 gridSize((width - 1)/blockSize.x + 1, (height - 1)/blockSize.y + 1);
    blurImgKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);
    hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
		  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
		  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    //Host copies result form divece memories
    CHECK(hipMemcpy(outPixels, d_outPixels, width * height * sizeof(uchar3), hipMemcpyDeviceToHost));
    //Host free device memories
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_outPixels));
    CHECK(hipFree(d_filter));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n",
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);

	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
